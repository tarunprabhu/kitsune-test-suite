/// Copyright 2009, Andrew Corrigan, acorriga@gmu.edu
// This code is from the AIAA-2009-4001 paper

#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <string>

struct Float3 {
  float x, y, z;
};

#define block_length 1

/*
 * Options
 *
 */
#define GAMMA 1.4
#define NDIM 3
#define NNB 4
#define RK 3 // 3rd order RK
#define ff_mach 1.2
#define deg_angle_of_attack 0.0f

/*
 * not options
 */
#define VAR_DENSITY 0
#define VAR_MOMENTUM 1
#define VAR_DENSITY_ENERGY (VAR_MOMENTUM + NDIM)
#define NVAR (VAR_DENSITY_ENERGY + 1)

void cpy(float *dst, const float *src, int N) {
  for (unsigned int i = 0; i < N; i++)
    dst[i] = src[i];
}

void dump(float *variables, int nel, int nelr, char *name) {
  std::string nameD = (std::string)name + "_density.dat";
  FILE *fpD = fopen(nameD.c_str(), "wb");
  fwrite(&variables[VAR_DENSITY * nelr], sizeof(float), nel, fpD);
  fclose(fpD);

  std::string nameM = (std::string)name + "_momentum.dat";
  FILE *fpM = fopen(nameM.c_str(), "wb");
  for (int j = 0; j < NDIM; j++)
    fwrite(&variables[(VAR_MOMENTUM + j) * nelr], sizeof(float), nel, fpM);
  fclose(fpM);

  std::string nameDE = (std::string)name + "_density_energy.dat";
  FILE *fpDE = fopen(nameDE.c_str(), "wb");
  fwrite(&variables[VAR_DENSITY_ENERGY * nelr], sizeof(float), nel, fpDE);
  fclose(fpDE);
}

void initialize_variables(int nelr, float *variables, float *ff_variable) {
  for (int i = 0; i < nelr; i++) {
    for (int j = 0; j < NVAR; j++)
      variables[i + j * nelr] = ff_variable[j];
  }
}

__host__ __device__ void compute_flux_contribution(
    const float density, const Float3 &momentum, const float density_energy,
    const float pressure, Float3 &velocity, Float3 &fc_momentum_x,
    Float3 &fc_momentum_y, Float3 &fc_momentum_z, Float3 &fc_density_energy) {
  fc_momentum_x.x = velocity.x * momentum.x + pressure;
  fc_momentum_x.y = velocity.x * momentum.y;
  fc_momentum_x.z = velocity.x * momentum.z;

  fc_momentum_y.x = fc_momentum_x.y;
  fc_momentum_y.y = velocity.y * momentum.y + pressure;
  fc_momentum_y.z = velocity.y * momentum.z;

  fc_momentum_z.x = fc_momentum_x.z;
  fc_momentum_z.y = fc_momentum_y.z;
  fc_momentum_z.z = velocity.z * momentum.z + pressure;

  float de_p = density_energy + pressure;
  fc_density_energy.x = velocity.x * de_p;
  fc_density_energy.y = velocity.y * de_p;
  fc_density_energy.z = velocity.z * de_p;
}

__device__ void compute_velocity(float density, const Float3 &momentum,
                                 Float3 &velocity) {
  velocity.x = momentum.x / density;
  velocity.y = momentum.y / density;
  velocity.z = momentum.z / density;
}

__device__ float compute_speed_sqd(const Float3 &velocity) {
  return velocity.x * velocity.x + velocity.y * velocity.y +
         velocity.z * velocity.z;
}

__device__ float compute_pressure(float density, float density_energy,
                                  float speed_sqd) {
  return (float(GAMMA) - float(1.0f)) *
         (density_energy - float(0.5f) * density * speed_sqd);
}

__device__ float compute_speed_of_sound(float density, float pressure) {
  return sqrtf(float(GAMMA) * pressure / density);
}

__global__ void compute_step_factor(int nelr, const float *variables,
                                    const float *areas, float *step_factors) {
  int blk = blockDim.x * blockIdx.x + threadIdx.x;
  if (blk < nelr / block_length) {
    int b_start = blk * block_length;
    int b_end =
        (blk + 1) * block_length > nelr ? nelr : (blk + 1) * block_length;

    for (int i = b_start; i < b_end; i++) {
      float density = variables[i + VAR_DENSITY * nelr];

      Float3 momentum;
      momentum.x = variables[i + (VAR_MOMENTUM + 0) * nelr];
      momentum.y = variables[i + (VAR_MOMENTUM + 1) * nelr];
      momentum.z = variables[i + (VAR_MOMENTUM + 2) * nelr];

      float density_energy = variables[i + VAR_DENSITY_ENERGY * nelr];
      Float3 velocity;
      compute_velocity(density, momentum, velocity);
      float speed_sqd = compute_speed_sqd(velocity);
      float pressure = compute_pressure(density, density_energy, speed_sqd);
      float speed_of_sound = compute_speed_of_sound(density, pressure);

      // dt = float(0.5f) * sqrt(areas[i]) / (||v|| + c).... but
      // when we do time stepping, this later would need to be divided
      // by the area, so we just do it all at once
      step_factors[i] =
          float(0.5f) / (sqrtf(areas[i]) * (sqrtf(speed_sqd) + speed_of_sound));
    }
  }
}

__global__ void compute_flux(int nelr, int *elements_surrounding_elements,
                             float *normals, float *variables, float *fluxes,
                             const float *ff_variable,
                             const Float3 ff_flux_contribution_momentum_x,
                             const Float3 ff_flux_contribution_momentum_y,
                             const Float3 ff_flux_contribution_momentum_z,
                             const Float3 ff_flux_contribution_density_energy) {
  using namespace std;
  const float smoothing_coefficient = 0.2f;

  int blk = blockDim.x * blockIdx.x + threadIdx.x;
  if (blk < nelr / block_length) {
    unsigned int b_start = blk * block_length;
    unsigned int b_end =
        (blk + 1) * block_length > nelr ? nelr : (blk + 1) * block_length;

    for (unsigned int i = b_start; i < b_end; ++i) {
      float density_i = variables[i + VAR_DENSITY * nelr];
      Float3 momentum_i;
      momentum_i.x = variables[i + (VAR_MOMENTUM + 0) * nelr];
      momentum_i.y = variables[i + (VAR_MOMENTUM + 1) * nelr];
      momentum_i.z = variables[i + (VAR_MOMENTUM + 2) * nelr];

      float density_energy_i = variables[i + VAR_DENSITY_ENERGY * nelr];

      Float3 velocity_i;
      compute_velocity(density_i, momentum_i, velocity_i);
      float speed_sqd_i = compute_speed_sqd(velocity_i);
      float speed_i = sqrtf(speed_sqd_i);
      float pressure_i =
          compute_pressure(density_i, density_energy_i, speed_sqd_i);
      float speed_of_sound_i = compute_speed_of_sound(density_i, pressure_i);
      Float3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y,
          flux_contribution_i_momentum_z;

      Float3 flux_contribution_i_density_energy;
      compute_flux_contribution(
          density_i, momentum_i, density_energy_i, pressure_i, velocity_i,
          flux_contribution_i_momentum_x, flux_contribution_i_momentum_y,
          flux_contribution_i_momentum_z, flux_contribution_i_density_energy);

      float flux_i_density = 0.0f;
      Float3 flux_i_momentum;
      flux_i_momentum.x = 0.0f;
      flux_i_momentum.y = 0.0f;
      flux_i_momentum.z = 0.0f;
      float flux_i_density_energy = 0.0f;

      Float3 velocity_nb;
      float density_nb, density_energy_nb;
      Float3 momentum_nb;
      Float3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y,
          flux_contribution_nb_momentum_z;
      Float3 flux_contribution_nb_density_energy;
      float speed_sqd_nb, speed_of_sound_nb, pressure_nb;

      for (int j = 0; j < NNB; j++) {
        Float3 normal;
        float normal_len;
        float factor;

        int nb = elements_surrounding_elements[i + j * nelr];
        normal.x = normals[i + (j + 0 * NNB) * nelr];
        normal.y = normals[i + (j + 1 * NNB) * nelr];
        normal.z = normals[i + (j + 2 * NNB) * nelr];
        normal_len = sqrtf(normal.x * normal.x + normal.y * normal.y +
                           normal.z * normal.z);

        if (nb >= 0) { // a legitimate neighbor
          density_nb = variables[nb + VAR_DENSITY * nelr];
          momentum_nb.x = variables[nb + (VAR_MOMENTUM)*nelr];
          momentum_nb.y = variables[nb + (VAR_MOMENTUM + 1) * nelr];
          momentum_nb.z = variables[nb + (VAR_MOMENTUM + 2) * nelr];
          density_energy_nb = variables[nb + VAR_DENSITY_ENERGY * nelr];
          compute_velocity(density_nb, momentum_nb, velocity_nb);
          speed_sqd_nb = compute_speed_sqd(velocity_nb);
          pressure_nb =
              compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
          speed_of_sound_nb = compute_speed_of_sound(density_nb, pressure_nb);
          compute_flux_contribution(
              density_nb, momentum_nb, density_energy_nb, pressure_nb,
              velocity_nb, flux_contribution_nb_momentum_x,
              flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z,
              flux_contribution_nb_density_energy);

          // artificial viscosity
          factor = -normal_len * smoothing_coefficient * 0.5f *
                   (speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i +
                    speed_of_sound_nb);
          flux_i_density += factor * (density_i - density_nb);
          flux_i_density_energy +=
              factor * (density_energy_i - density_energy_nb);
          flux_i_momentum.x += factor * (momentum_i.x - momentum_nb.x);
          flux_i_momentum.y += factor * (momentum_i.y - momentum_nb.y);
          flux_i_momentum.z += factor * (momentum_i.z - momentum_nb.z);

          // accumulate cell-centered fluxes
          factor = 0.5f * normal.x;
          flux_i_density += factor * (momentum_nb.x + momentum_i.x);
          flux_i_density_energy +=
              factor * (flux_contribution_nb_density_energy.x +
                        flux_contribution_i_density_energy.x);
          flux_i_momentum.x += factor * (flux_contribution_nb_momentum_x.x +
                                         flux_contribution_i_momentum_x.x);
          flux_i_momentum.y += factor * (flux_contribution_nb_momentum_y.x +
                                         flux_contribution_i_momentum_y.x);
          flux_i_momentum.z += factor * (flux_contribution_nb_momentum_z.x +
                                         flux_contribution_i_momentum_z.x);

          factor = 0.5f * normal.y;
          flux_i_density += factor * (momentum_nb.y + momentum_i.y);
          flux_i_density_energy +=
              factor * (flux_contribution_nb_density_energy.y +
                        flux_contribution_i_density_energy.y);
          flux_i_momentum.x += factor * (flux_contribution_nb_momentum_x.y +
                                         flux_contribution_i_momentum_x.y);
          flux_i_momentum.y += factor * (flux_contribution_nb_momentum_y.y +
                                         flux_contribution_i_momentum_y.y);
          flux_i_momentum.z += factor * (flux_contribution_nb_momentum_z.y +
                                         flux_contribution_i_momentum_z.y);

          factor = 0.5f * normal.z;
          flux_i_density += factor * (momentum_nb.z + momentum_i.z);
          flux_i_density_energy +=
              factor * (flux_contribution_nb_density_energy.z +
                        flux_contribution_i_density_energy.z);
          flux_i_momentum.x += factor * (flux_contribution_nb_momentum_x.z +
                                         flux_contribution_i_momentum_x.z);
          flux_i_momentum.y += factor * (flux_contribution_nb_momentum_y.z +
                                         flux_contribution_i_momentum_y.z);
          flux_i_momentum.z += factor * (flux_contribution_nb_momentum_z.z +
                                         flux_contribution_i_momentum_z.z);
        } else if (nb == -1) { // a wing boundary
          flux_i_momentum.x += normal.x * pressure_i;
          flux_i_momentum.y += normal.y * pressure_i;
          flux_i_momentum.z += normal.z * pressure_i;
        } else if (nb == -2) { // a far field boundary
          factor = 0.5f * normal.x;
          flux_i_density += factor * (ff_variable[VAR_MOMENTUM] + momentum_i.x);
          flux_i_density_energy +=
              factor * (ff_flux_contribution_density_energy.x +
                        flux_contribution_i_density_energy.x);
          flux_i_momentum.x += factor * (ff_flux_contribution_momentum_x.x +
                                         flux_contribution_i_momentum_x.x);
          flux_i_momentum.y += factor * (ff_flux_contribution_momentum_y.x +
                                         flux_contribution_i_momentum_y.x);
          flux_i_momentum.z += factor * (ff_flux_contribution_momentum_z.x +
                                         flux_contribution_i_momentum_z.x);

          factor = float(0.5f) * normal.y;
          flux_i_density +=
              factor * (ff_variable[VAR_MOMENTUM + 1] + momentum_i.y);
          flux_i_density_energy +=
              factor * (ff_flux_contribution_density_energy.y +
                        flux_contribution_i_density_energy.y);
          flux_i_momentum.x += factor * (ff_flux_contribution_momentum_x.y +
                                         flux_contribution_i_momentum_x.y);
          flux_i_momentum.y += factor * (ff_flux_contribution_momentum_y.y +
                                         flux_contribution_i_momentum_y.y);
          flux_i_momentum.z += factor * (ff_flux_contribution_momentum_z.y +
                                         flux_contribution_i_momentum_z.y);

          factor = float(0.5f) * normal.z;
          flux_i_density +=
              factor * (ff_variable[VAR_MOMENTUM + 2] + momentum_i.z);
          flux_i_density_energy +=
              factor * (ff_flux_contribution_density_energy.z +
                        flux_contribution_i_density_energy.z);
          flux_i_momentum.x += factor * (ff_flux_contribution_momentum_x.z +
                                         flux_contribution_i_momentum_x.z);
          flux_i_momentum.y += factor * (ff_flux_contribution_momentum_y.z +
                                         flux_contribution_i_momentum_y.z);
          flux_i_momentum.z += factor * (ff_flux_contribution_momentum_z.z +
                                         flux_contribution_i_momentum_z.z);
        }
      }

      fluxes[i + VAR_DENSITY * nelr] = flux_i_density;
      fluxes[i + (VAR_MOMENTUM + 0) * nelr] = flux_i_momentum.x;
      fluxes[i + (VAR_MOMENTUM + 1) * nelr] = flux_i_momentum.y;
      fluxes[i + (VAR_MOMENTUM + 2) * nelr] = flux_i_momentum.z;
      fluxes[i + VAR_DENSITY_ENERGY * nelr] = flux_i_density_energy;
    }
  }
}

__global__ void time_step(int j, int nelr, float *old_variables,
                          float *variables, float *step_factors,
                          float *fluxes) {
  int blk = blockDim.x * blockIdx.x + threadIdx.x;
  if (blk < nelr / block_length) {
    int b_start = blk * block_length;
    int b_end =
        (blk + 1) * block_length > nelr ? nelr : (blk + 1) * block_length;
    for (int i = b_start; i < b_end; ++i) {
      float factor = step_factors[i] / float(RK + 1 - j);
      variables[i + VAR_DENSITY * nelr] =
          old_variables[i + VAR_DENSITY * nelr] +
          factor * fluxes[i + VAR_DENSITY * nelr];
      variables[i + (VAR_MOMENTUM + 0) * nelr] =
          old_variables[i + (VAR_MOMENTUM + 0) * nelr] +
          factor * fluxes[i + (VAR_MOMENTUM + 0) * nelr];
      variables[i + (VAR_MOMENTUM + 1) * nelr] =
          old_variables[i + (VAR_MOMENTUM + 1) * nelr] +
          factor * fluxes[i + (VAR_MOMENTUM + 1) * nelr];
      variables[i + (VAR_MOMENTUM + 2) * nelr] =
          old_variables[i + (VAR_MOMENTUM + 2) * nelr] +
          factor * fluxes[i + (VAR_MOMENTUM + 2) * nelr];
      variables[i + VAR_DENSITY_ENERGY * nelr] =
          old_variables[i + VAR_DENSITY_ENERGY * nelr] +
          factor * fluxes[i + VAR_DENSITY_ENERGY * nelr];
    }
  }
}

/*
 * Main function
 */
int main(int argc, char **argv) {
  using namespace std;

  if (argc < 2) {
    cout << "specify data file name" << endl;
    return 0;
  }

  int iterations = 4000;
  if (argc > 2)
    iterations = atoi(argv[2]);

  const char *data_file_name = argv[1];

  cout << setprecision(5);
  cout << "\n";
  cout << "---- euler3d benchmark (cuda) ----\n\n"
       << "  Input file : " << data_file_name << "\n"
       << "  Iterations : " << iterations << ".\n\n";
  cout << "  Reading input data, allocating arrays, initializing data, etc..."
       << std::flush;
  auto total_start_time = chrono::steady_clock::now();

  // these need to be computed the first time in order to compute time step
  hipError_t err = hipSuccess;
  float *ff_variable;
  err = hipMallocManaged(&ff_variable, NVAR * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory!\n");
    return 1;
  }
  Float3 ff_flux_contribution_momentum_x, ff_flux_contribution_momentum_y,
      ff_flux_contribution_momentum_z;
  Float3 ff_flux_contribution_density_energy;

  // set far field conditions
  const float angle_of_attack =
      float(3.1415926535897931 / 180.0f) * float(deg_angle_of_attack);

  ff_variable[VAR_DENSITY] = float(1.4);

  float ff_pressure = float(1.0f);
  float ff_speed_of_sound =
      sqrtf(GAMMA * ff_pressure / ff_variable[VAR_DENSITY]);
  float ff_speed = float(ff_mach) * ff_speed_of_sound;

  Float3 ff_velocity;
  ff_velocity.x = ff_speed * float(cosf((float)angle_of_attack));
  ff_velocity.y = ff_speed * float(sinf((float)angle_of_attack));
  ff_velocity.z = 0.0f;

  ff_variable[VAR_MOMENTUM + 0] = ff_variable[VAR_DENSITY] * ff_velocity.x;
  ff_variable[VAR_MOMENTUM + 1] = ff_variable[VAR_DENSITY] * ff_velocity.y;
  ff_variable[VAR_MOMENTUM + 2] = ff_variable[VAR_DENSITY] * ff_velocity.z;

  ff_variable[VAR_DENSITY_ENERGY] =
      ff_variable[VAR_DENSITY] * (float(0.5f) * (ff_speed * ff_speed)) +
      (ff_pressure / float(GAMMA - 1.0f));

  Float3 ff_momentum;
  ff_momentum.x = *(ff_variable + VAR_MOMENTUM + 0);
  ff_momentum.y = *(ff_variable + VAR_MOMENTUM + 1);
  ff_momentum.z = *(ff_variable + VAR_MOMENTUM + 2);
  compute_flux_contribution(
      ff_variable[VAR_DENSITY], ff_momentum, ff_variable[VAR_DENSITY_ENERGY],
      ff_pressure, ff_velocity, ff_flux_contribution_momentum_x,
      ff_flux_contribution_momentum_y, ff_flux_contribution_momentum_z,
      ff_flux_contribution_density_energy);

  int nel;
  int nelr;

  // read in domain geometry
  float *areas;
  int *elements_surrounding_elements;
  float *normals;

  ifstream file(data_file_name);
  file >> nel;
  nelr = block_length * ((nel / block_length) + min(1, nel % block_length));

  err = hipMallocManaged(&areas, nelr * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory!\n");
    return 1;
  }
  err = hipMallocManaged(&elements_surrounding_elements,
                          nelr * NNB * sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory!\n");
    return 1;
  }
  err = hipMallocManaged(&normals, NDIM * NNB * nelr * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory!\n");
    return 1;
  }

  // read in data
  for (int i = 0; i < nel; i++) {
    file >> areas[i];
    for (int j = 0; j < NNB; j++) {
      file >> elements_surrounding_elements[i + j * nelr];
      if (elements_surrounding_elements[i + j * nelr] < 0)
        elements_surrounding_elements[i + j * nelr] = -1;
      // it's coming in with Fortran numbering
      elements_surrounding_elements[i + j * nelr]--;

      for (int k = 0; k < NDIM; k++) {
        file >> normals[i + (j + k * NNB) * nelr];
        normals[i + (j + k * NNB) * nelr] = -normals[i + (j + k * NNB) * nelr];
      }
    }
  }

  // fill in remaining data
  int last = nel - 1;
  for (int i = nel; i < nelr; i++) {
    areas[i] = areas[last];
    for (int j = 0; j < NNB; j++) {
      // duplicate the last element
      elements_surrounding_elements[i + j * nelr] =
          elements_surrounding_elements[last + j * nelr];
      for (int k = 0; k < NDIM; k++)
        normals[i + (j + k * NNB) * nelr] =
            normals[last + (j + k * NNB) * nelr];
    }
  }

  // Create arrays and set initial conditions
  float *variables;
  err = hipMallocManaged(&variables, nelr * NVAR * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory!\n");
    return 1;
  }
  cout << "  done.\n\n";

  cout << "  Starting benchmark...\n" << std::flush;
  auto start_time = chrono::steady_clock::now();
  int threadsPerBlock = 256;
  int blocksPerGrid =
      (nelr / block_length + threadsPerBlock - 1) / threadsPerBlock;

  initialize_variables(nelr, variables, ff_variable);
  float *old_variables;
  err = hipMallocManaged(&old_variables, nelr * NVAR * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory!\n");
    return 1;
  }
  float *fluxes;
  err = hipMallocManaged(&fluxes, nelr * NVAR * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory!\n");
    return 1;
  }
  float *step_factors;
  err = hipMallocManaged(&step_factors, nelr * sizeof(float));
  if (err != hipSuccess) {
    fprintf(stderr, "failed to allocate managed memory!\n");
    return 1;
  }

  double *rk_times = new double[iterations];

  // Begin iterations
  double copy_total = 0.0;
  double sf_total = 0.0;
  double rk_total = 0.0;

  for (int i = 0; i < iterations; i++) {
    auto copy_start = chrono::steady_clock::now();
    cpy(old_variables, variables, nelr * NVAR);
    auto copy_end = chrono::steady_clock::now();
    double time = chrono::duration<double>(copy_end - copy_start).count();
    copy_total += time;

    // for the first iteration we compute the time step
    auto sf_start = chrono::steady_clock::now();
    compute_step_factor<<<blocksPerGrid, threadsPerBlock>>>(
        nelr, variables, areas, step_factors);
    hipDeviceSynchronize();
    auto sf_end = chrono::steady_clock::now();
    time = chrono::duration<double>(sf_end - sf_start).count();
    sf_total += time;

    auto rk_start = chrono::steady_clock::now();
    for (int j = 0; j < RK; j++) {
      compute_flux<<<blocksPerGrid, threadsPerBlock>>>(
          nelr, elements_surrounding_elements, normals, variables, fluxes,
          ff_variable, ff_flux_contribution_momentum_x,
          ff_flux_contribution_momentum_y, ff_flux_contribution_momentum_z,
          ff_flux_contribution_density_energy);
      hipDeviceSynchronize();
      time_step<<<blocksPerGrid, threadsPerBlock>>>(
          j, nelr, old_variables, variables, step_factors, fluxes);
      hipDeviceSynchronize();
    }
    auto rk_end = chrono::steady_clock::now();
    time = chrono::duration<double>(rk_end - rk_start).count();
    rk_times[i] = time;
    rk_total += time;
  }
  auto end_time = chrono::steady_clock::now();
  double elapsed_time = chrono::duration<double>(end_time - start_time).count();

  dump(variables, nel, nelr, argv[0]);

  end_time = chrono::steady_clock::now();
  double total_time =
      chrono::duration<double>(end_time - total_start_time).count();
  double rk_mean = rk_total / (iterations - 1);
  double sum = 0.0;
  for (int i = 1; i < iterations; i++) {
    double dist = rk_times[i] - rk_mean;
    sum += dist * dist;
  }
  double rk_std_dev = sqrt(sum / iterations);

  cout << "\n"
       << "      Total time : " << total_time << " seconds.\n"
       << "    Compute time : " << elapsed_time << " seconds.\n"
       << "            copy : " << copy_total
       << " seconds (average: " << copy_total / iterations << " seconds).\n"
       << "              sf : " << sf_total
       << " seconds (average: " << sf_total / iterations << " seconds).\n"
       << "              rk : " << rk_total << " seconds (average: " << rk_mean
       << " seconds / std dev:" << rk_std_dev << ").\n"
       << "*** " << elapsed_time << ", " << elapsed_time << "\n"
       << "----\n\n";

  return 0;
}
